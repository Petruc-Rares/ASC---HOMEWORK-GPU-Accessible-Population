#include "hip/hip_runtime.h"
#include <fstream>
#include <vector>
#include <string>
#include <iostream>
#include <math.h>

#include "helper.h"
#define BLOCK_SIZE 256

unsigned int fill_data(std::vector<float>& longit, std::vector <float>& lat, std::vector<unsigned int>& pop, char *input_filename) {
        std::ifstream ifs(input_filename);

        std::string aux_string;
        float aux_longit;
        float aux_lat;
        unsigned int aux_pop;

        unsigned int no_lines = 0;

        while (ifs >> aux_string >> aux_longit >> aux_lat >> aux_pop) {
                //printf("shoul enter here\n");
                longit.push_back(aux_longit);
                lat.push_back(aux_lat);
                pop.push_back(aux_pop);
                no_lines++;
        }

        ifs.close();
        return no_lines;
}


__global__ void master(unsigned int *results, float *longits, float *lats, unsigned int *pops, unsigned int file_size, float km_range) {
        unsigned int idx =  blockIdx.x * blockDim.x + threadIdx.x;
        unsigned int my_file_size = file_size;
        if (idx >= file_size) return;

        float my_degree_to_radians = DEGREE_TO_RADIANS;
        float ninety_degrees = 90.f;
        float sin_2_cos_2 = 1.0f;

        float sin_phi1 = sin((ninety_degrees - lats[idx]) * my_degree_to_radians);
        float cos_phi1 = sqrt(sin_2_cos_2 - sin_phi1 * sin_phi1);

        float theta1 = longits[idx] * my_degree_to_radians;

        float my_km_range = km_range;
        unsigned int my_pop = pops[idx];

        float constant_dec = ninety_degrees * my_degree_to_radians;

        for (unsigned int i = idx + 1; i < my_file_size; i++) {

                float sin_phi2 = sin(constant_dec - lats[i] * my_degree_to_radians);
                float cos_phi2 = sqrt(sin_2_cos_2 - sin_phi2 * sin_phi2);

                float theta2 = longits[i] * my_degree_to_radians;

                float cs = sin_phi1 * sin_phi2 * cos(theta1 - theta2) + cos_phi1 * cos_phi2;
                if (cs > 1) {
                        cs = 1;
                } else if (cs < -1) {
                        cs = -1;
                }

                if (6371.f * acos(cs) <= my_km_range) {
                        atomicAdd(&results[idx], pops[i]);
                        atomicAdd(&results[i], my_pop);
                }

        }
}

void writeResults(char *output_file_name, unsigned int *results, unsigned int size) {
        std::ofstream ofs(output_file_name);

        for (int i = 0; i < size; i++) {
               ofs << results[i] << "\n";
        }

        ofs.close();
}

int main(int argc, char *argv[]) {
        if (argc == 1) {
                std::cout << "Usage: ./gpu_my_sol <kmrange1> <file1in> <file1out> ..." << std::endl;
        } else if ((argc - 1) % 3 != 0) {
                std::cout << "Usage: ./gpu_my_sol <kmrange1> <file1in> <file1out> ,,," << std::endl;
        }

        for(int argcID = 1; argcID < argc; argcID += 3) {
                std::vector<float> longit(0);
                std::vector<float> lat(0);
                std::vector<unsigned int> pop(0);
                unsigned int *no_lines_file = 0;
                float *km_range = 0;

                float *longits = 0;
                float *lats = 0;
                unsigned int *pops = 0;
                unsigned int *results = 0;

                hipMallocManaged(&no_lines_file, sizeof(unsigned int));
                hipMallocManaged(&km_range, sizeof(float));

                *km_range = atof(argv[argcID]);
                *no_lines_file = fill_data(longit, lat, pop, argv[argcID + 1]);

                hipMallocManaged(&longits, *no_lines_file * sizeof(float));              
                hipMallocManaged(&lats, *no_lines_file * sizeof(float));
                hipMallocManaged(&pops, *no_lines_file * sizeof(unsigned int));
                hipMallocManaged(&results, *no_lines_file * sizeof(unsigned int));
        
                // reading done    
                hipMemcpy(longits, longit.data(), *no_lines_file * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(lats, lat.data(), *no_lines_file * sizeof(float), hipMemcpyHostToDevice);
                hipMemcpy(pops, pop.data(), *no_lines_file * sizeof(unsigned int), hipMemcpyHostToDevice);
                hipMemcpy(results, pop.data(), *no_lines_file * sizeof(unsigned int), hipMemcpyHostToDevice);

                unsigned int block_no = *no_lines_file / BLOCK_SIZE;
                if (*no_lines_file % BLOCK_SIZE) {
                        block_no++;
                }


                master<<<block_no, BLOCK_SIZE>>>(results, longits, lats, pops, *no_lines_file, *km_range);
                if (hipSuccess != hipGetLastError()) {
                        printf("pisici\n");
                        return 1;
                }

                // wait for parent to complete
                if (hipSuccess != hipDeviceSynchronize()) {
                        printf("caini\n");
                        return 2;
                }
                
                writeResults(argv[argcID + 2], results, *no_lines_file);
        
                hipFree(no_lines_file);
                hipFree(km_range);
                hipFree(longits);
                hipFree(lats);
                hipFree(pops);
                hipFree(results);
        }  
}